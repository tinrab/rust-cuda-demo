#include "helpers.h"
#include "lib_cublas.h"

hipError_t cublas_sgemm(
    int n, const float* a, const float* b, float* c, float alpha, float beta,
    hipblasStatus_t* cublas_status
) {
    int n2 = n * n;
    // Device memory
    float* device_a = 0;
    float* device_b = 0;
    float* device_c = 0;

    hipblasHandle_t handle;
    *cublas_status = hipblasCreate(&handle);
    if (*cublas_status != HIPBLAS_STATUS_SUCCESS) {
        return hipErrorUnknown;
    }

    // Allocate device memory
    CHECK_CUDA(hipMalloc(
        reinterpret_cast<void**>(&device_a), n2 * sizeof(device_a[0])
    ));
    CHECK_CUDA(hipMalloc(
        reinterpret_cast<void**>(&device_b), n2 * sizeof(device_b[0])
    ));
    CHECK_CUDA(hipMalloc(
        reinterpret_cast<void**>(&device_c), n2 * sizeof(device_c[0])
    ));

    // Copy data from host to device
    *cublas_status = hipblasSetVector(n2, sizeof(a[0]), a, 1, device_a, 1);
    if (*cublas_status != HIPBLAS_STATUS_SUCCESS) {
        return hipErrorUnknown;
    }
    *cublas_status = hipblasSetVector(n2, sizeof(b[0]), b, 1, device_b, 1);
    if (*cublas_status != HIPBLAS_STATUS_SUCCESS) {
        return hipErrorUnknown;
    }
    *cublas_status = hipblasSetVector(n2, sizeof(c[0]), c, 1, device_c, 1);
    if (*cublas_status != HIPBLAS_STATUS_SUCCESS) {
        return hipErrorUnknown;
    }

    // Perform matrix multiplication
    *cublas_status = hipblasSgemm(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, device_a, n,
        device_b, n, &beta, device_c, n
    );
    if (*cublas_status != HIPBLAS_STATUS_SUCCESS) {
        return hipErrorUnknown;
    }

    // Copy data from device to host
    *cublas_status = hipblasGetVector(n2, sizeof(c[0]), device_c, 1, c, 1);
    if (*cublas_status != HIPBLAS_STATUS_SUCCESS) {
        return hipErrorUnknown;
    }

    // Free resources
    CHECK_CUDA(hipFree(device_a));
    CHECK_CUDA(hipFree(device_b));
    CHECK_CUDA(hipFree(device_c));

    *cublas_status = hipblasDestroy(handle);
    if (*cublas_status != HIPBLAS_STATUS_SUCCESS) {
        return hipErrorUnknown;
    }

    return hipSuccess;
}
