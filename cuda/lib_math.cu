#include "hip/hip_runtime.h"
#include "helpers.h"
#include "lib_math.h"

__global__ void vector_add_kernel(
    int n, const float* a, const float* b, float* c
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

hipError_t math_vector_add(int n, const float* a, const float* b, float* c) {
    const size_t BYTES = n * sizeof(float);

    float* device_a = 0;
    float* device_b = 0;
    float* device_c = 0;
    CHECK_CUDA(hipMalloc(reinterpret_cast<void**>(&device_a), BYTES));
    CHECK_CUDA(hipMalloc(reinterpret_cast<void**>(&device_b), BYTES));
    CHECK_CUDA(hipMalloc(reinterpret_cast<void**>(&device_c), BYTES));

    CHECK_CUDA(hipMemcpy(device_a, a, BYTES, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(device_b, b, BYTES, hipMemcpyHostToDevice));

    vector_add_kernel<<<(n + 255) / 256, 256>>>(
        n, device_a, device_b, device_c
    );
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipGetLastError());

    CHECK_CUDA(hipMemcpy(c, device_c, BYTES, hipMemcpyDeviceToHost));

    CHECK_CUDA(hipFree(device_a));
    CHECK_CUDA(hipFree(device_b));
    CHECK_CUDA(hipFree(device_c));

    return hipSuccess;
}
