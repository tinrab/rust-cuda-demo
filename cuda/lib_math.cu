#include "hip/hip_runtime.h"
#include "helpers.h"
#include "lib_math.h"

__global__ void vector_add_kernel(
    int n,
    const float* a,
    const float* b,
    float* c
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

hipError_t math_vector_add(int n, const float* a, const float* b, float* c) {
    const size_t BYTES = n * sizeof(float);

    float* device_a = 0;
    float* device_b = 0;
    float* device_c = 0;
    CHECK_CUDA(hipMalloc(reinterpret_cast<void**>(&device_a), BYTES));
    CHECK_CUDA(hipMalloc(reinterpret_cast<void**>(&device_b), BYTES));
    CHECK_CUDA(hipMalloc(reinterpret_cast<void**>(&device_c), BYTES));

    CHECK_CUDA(hipMemcpy(device_a, a, BYTES, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(device_b, b, BYTES, hipMemcpyHostToDevice));

    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    dim3 grid(blocks, 1, 1);
    dim3 block(threads, 1, 1);

    vector_add_kernel<<<grid, block>>>(n, device_a, device_b, device_c);

    // void* args[] = {
    //     reinterpret_cast<void*>(&n),
    //     reinterpret_cast<void*>(&device_a),
    //     reinterpret_cast<void*>(&device_b),
    //     reinterpret_cast<void*>(&device_c),
    // };
    // hipLaunchKernel((void*)vector_add_kernel, grid, block, args);

    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipGetLastError());

    CHECK_CUDA(hipMemcpy(c, device_c, BYTES, hipMemcpyDeviceToHost));

    CHECK_CUDA(hipFree(device_a));
    CHECK_CUDA(hipFree(device_b));
    CHECK_CUDA(hipFree(device_c));

    return hipSuccess;
}
